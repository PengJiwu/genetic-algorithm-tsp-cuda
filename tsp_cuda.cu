#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "cuda_utils.h"

#include "tsp.h"
#include "tsp_cuda.h"
#include "warmup.h"

__global__ void tsp_solver(unsigned int *s, unsigned int *d, float *ps, float *pd, int *found)
{
	/* temporarily writing a dummy kernel that just copies s into d */
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	d[tid] = s[tid];
}

int run(city * cities, int N, int maxgenerations, int maxpopulation, float optimal, unsigned int * result_tour)
{
	unsigned int *s1, *tour;
	float *p1;
	/* s1 is actually a 2-D matrix modeled as 1-D array. Its rows represent each individual (a tour).
	 * and the elements in row represents each city. Thus, each row contain N unique elements from 0..N-1
	 * which represent the city index to be visited. This index corresponds with the original "cities"
	 * list. e.g. if s1[i*N + 0] = 3 and s1[i*N + 1] = 5, then it means that the ith tour starts with city
	 * represented by cities[3] and the next city visited is cities[5] and so on..
	 * The number of such rows (the values of i) = maxpopulation */
	s1 = (unsigned int *)  malloc(N * maxpopulation * sizeof(unsigned int));

	/* p1 is the tour length of each tour in s1. Thus, the number of elements in p1 = maxpopulation */
	p1 = (float *) malloc(maxpopulation * sizeof(float));

	int i,j;
	/* initialize population with random tours */
	for(i = 0 ; i < maxpopulation ; i++)
	{
		tour = (s1 + i*N);
		generate_random_tour(tour, N);
		p1[i] = tour_length(tour, N, cities);
		if(optimal >= 0)	/* an optimal solution has been provided. */
		{
			if(is_acceptable(tour, N, cities, optimal))
			{
				for(j = 0 ; j < N ; j++)
					result_tour[j] = tour[j];
				return 0;
			}
		}
	}

	/* Now initialize the device memory. These are corresponding device pointers for s1 and p1.
	 * We have two copies of them so that we can deal with 2 generations at a time (parents and their offsprings).
	 * These would be used similar to double buffering so that they alternate between every generation.
	 * e.g. d_s1 have parents while d_s2 have their offsprings/mutations during 1st gen; 
	 *      d_s2 have parents while d_s1 have their offsprings/mutations during 2nd gen; and so on..
	 */
	unsigned int *d_s1, *d_s2, *d_stemp;
	float *d_p1, *d_p2, *d_ptemp;
	int h_found = -1, *d_found;		/* the device updates this to notify the host that acceptable solution was found */

	CUDA_CHECK_ERROR(  hipMalloc(&d_s1, N * maxpopulation * sizeof(unsigned int)) );
	CUDA_CHECK_ERROR(  hipMalloc(&d_s2, N * maxpopulation * sizeof(unsigned int)) );
	CUDA_CHECK_ERROR(  hipMalloc(&d_p1, maxpopulation * sizeof(float)) );
	CUDA_CHECK_ERROR(  hipMalloc(&d_p2, maxpopulation * sizeof(float)) );
	
	CUDA_CHECK_ERROR(  hipMemcpy(d_s1,  s1,   N * maxpopulation * sizeof(unsigned int),  hipMemcpyHostToDevice) );
	CUDA_CHECK_ERROR(  hipMemcpy(d_p1,  p1,   maxpopulation * sizeof(float),  hipMemcpyHostToDevice) );

	CUDA_CHECK_ERROR(  hipMalloc(&d_found, sizeof(int)) );
	CUDA_CHECK_ERROR(  hipMemcpy(d_found,  &h_found, sizeof(int),  hipMemcpyHostToDevice) );

	dim3 grid(maxpopulation);
	dim3 block(N);

	/* warm-up */
	warm_up(maxpopulation, N);

	/* XXX : execute the core genetic algorithm loop to solve TSP */
	int generation = 0;
	while(generation < maxgenerations)
	{
		tsp_solver<<< grid, block >>> (d_s1, d_s2, d_p1, d_p2, d_found);
		CUDA_CHECK_ERROR( hipMemcpy( &h_found, d_found, sizeof(int), hipMemcpyDeviceToHost) );
		if(h_found > 0)
			break;

		/* swap d_s1 and d_s2 */
		d_stemp = d_s1;
		d_s1 = d_s2;
		d_s2 = d_stemp;

		/* swap d_p1 and d_p2 */
		d_ptemp = d_p1;
		d_p1 = d_p2;
		d_p2 = d_ptemp;

		/* increment the generation */
		generation++;
	}

	if(h_found > 0)
		fprintf(stderr, "Optimal solution was found\n");
	else
		fprintf(stderr, "Optimal solution was NOT found\n");

	/* Copy back the results in any case */
	CUDA_CHECK_ERROR( hipMemcpy( s1, d_s2, N * maxpopulation * sizeof(unsigned int), hipMemcpyDeviceToHost) );
	CUDA_CHECK_ERROR( hipMemcpy( p1, d_p2, maxpopulation * sizeof(unsigned int), hipMemcpyDeviceToHost) );

	/* clean up device memory */
	CUDA_CHECK_ERROR( hipFree(d_s1) );
	CUDA_CHECK_ERROR( hipFree(d_s2) );
	CUDA_CHECK_ERROR( hipFree(d_p1) );
	CUDA_CHECK_ERROR( hipFree(d_p2) );
	CUDA_CHECK_ERROR( hipFree(d_found) );

	/* send the result back to driver */
	if(h_found > 0)
	{
		for(j = 0 ; j < N ; j++)
			result_tour[j] = (s1 + h_found * N)[j];
	}
	else
	{
		/* Optimal solution was not found.
		 * TODO : handle this effectively and send so far best tour found,
		 *	  currently returning the zeroth tour */
		for(j = 0 ; j < N ; j++)
			result_tour[j] = s1[j];
	}

	return generation;
}

void generate_random_tour(unsigned int *tour, int size)
{
	/* Each tour should have 'size' number of unique elements in some random sequence.
	 * We use different seeds for different tours so than the pseudo-random number
	 * generator will not have same results for all tours */

	/* set seed depending on current time (number of seconds since epoch) */
	srandom(time(NULL));
	
	/* generate (pseudo-)random tour */
	int i;
	unsigned int r;
	for(i = 0 ; i < size ; i++)
	{
		r = random() % size;
		while(is_present(tour, r, i))			/* TODO : pretty crude way of generating unique numbers in 0..N. */
			r = random() % size;
		tour[i] = r;
	}
}

int is_present(unsigned int * haystack, unsigned int needle, int size)		// This is simple linear search. TODO : binary search if reqd.
{
	int i;
	for(i = 0 ; i < size ; i++)
	{
		if(haystack[i] == needle)
			return 1;
	}
	return 0;
}

int is_acceptable(unsigned int * tour, int N, city * cities, float optimal)
{
	float len = tour_length(tour, N, cities);
	float delta = len - optimal;
	delta = (delta < 0) ? -delta : delta;
	float allowed_err = optimal * PERCENT_ERROR / 100.0;
	if(delta < allowed_err)
		return 1;
	else
		return 0;
}
